#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <vector>

__global__ void node_pos_to_pin_pos_cuda_kernel(
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> node_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> pin_id2node_id,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> pin_pos,
    int num_pins) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // pin index
    if (i < num_pins) {
        const int c = index & 1;  // channel index
        int64_t node_id = pin_id2node_id[i];
        pin_pos[i][c] += node_pos[node_id][c];
    }
}

__global__ void calc_node_grad_deterministic_cuda_kernel(
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> node_grad,
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> pin_grad,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> node2pin_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> node2pin_list_end,
    int num_nodes) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // node index
    if (i < num_nodes) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = node2pin_list_end[i - 1];
        }
        int64_t end_idx = node2pin_list_end[i];
        if (end_idx != start_idx) {
            node_grad[i][c] += pin_grad[node2pin_list[start_idx]][c];
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                node_grad[i][c] += pin_grad[node2pin_list[idx]][c];
            }
        }
    }
}

__global__ void wa_wirelength_pin_root_timing_kernel(
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> pin_pos,
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> timing_pin_weight,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list_end,
    const torch::PackedTensorAccessor32<bool, 1, torch::RestrictPtrTraits> net_mask,
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> net_weight,
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> hpwl_scale,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> partial_wa_wl,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> partial_hpwl,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> pin_grad,
    int num_nets,
    float inv_gamma) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // net index
    if (i < num_nets && net_mask[i]) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = hyperedge_list_end[i - 1];
        }
        int64_t end_idx = hyperedge_list_end[i];
        if (end_idx != start_idx) {
            int64_t root_id = hyperedge_list[start_idx];
            float x_min = pin_pos[root_id][c];
            float x_max = pin_pos[root_id][c];
            float root_x = pin_pos[root_id][c];
            float recenter_exp_r = exp((root_x - x_max) * inv_gamma);
            float recenter_exp_nr = exp((x_min - root_x) * inv_gamma);
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                float cur_x = pin_pos[hyperedge_list[idx]][c];
                x_min = min(cur_x, x_min);
                x_max = max(cur_x, x_max);
            }
            partial_hpwl[i][c] = round((x_max - x_min) * hpwl_scale[c]);

            float sum_x_exp_x = root_x * recenter_exp_r;
            float sum_x_exp_nx = root_x * recenter_exp_nr;
            float sum_exp_x = recenter_exp_r;
            float sum_exp_nx = recenter_exp_nr;
            float wl_sum_x_exp_x = sum_x_exp_x;
            float wl_sum_x_exp_nx = sum_x_exp_nx;
            float wl_sum_exp_x = sum_exp_x;
            float wl_sum_exp_nx = sum_exp_nx;
            // pin-root gradient
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                int64_t pin_id = hyperedge_list[idx];
                float cur_x = pin_pos[pin_id][c];
                float recenter_exp_x = exp((cur_x - x_max) * inv_gamma);
                float recenter_exp_nx = exp((x_min - cur_x) * inv_gamma);

                float sum_x_exp_x = cur_x * recenter_exp_x + root_x * recenter_exp_r;
                float sum_x_exp_nx = cur_x * recenter_exp_nx + root_x * recenter_exp_nr;
                float sum_exp_x = recenter_exp_x + recenter_exp_r;
                float sum_exp_nx = recenter_exp_nx + recenter_exp_nr;
                wl_sum_x_exp_x += cur_x * recenter_exp_x;
                wl_sum_x_exp_nx += cur_x * recenter_exp_nx;
                wl_sum_exp_x += recenter_exp_x;
                wl_sum_exp_nx += recenter_exp_nx;

                float inv_sum_exp_x = 1 / sum_exp_x;
                float inv_sum_exp_nx = 1 / sum_exp_nx;
                float s_x = sum_x_exp_x * inv_sum_exp_x;
                float ns_nx = sum_x_exp_nx * inv_sum_exp_nx;
                partial_wa_wl[i][c] += s_x - ns_nx;
                float x_coeff = inv_gamma * inv_sum_exp_x;
                float nx_coeff = -inv_gamma * inv_sum_exp_nx;
                float grad_const = (1 - inv_gamma * s_x) * inv_sum_exp_x;
                float grad_nconst = (1 + inv_gamma * ns_nx) * inv_sum_exp_nx;

                float x_grad = (grad_const + x_coeff * cur_x) * recenter_exp_x -
                               (grad_nconst + nx_coeff * cur_x) * recenter_exp_nx;
                float root_grad = (grad_const + x_coeff * root_x) * recenter_exp_r -
                                  (grad_nconst + nx_coeff * root_x) * recenter_exp_nr;

                float delta_x = timing_pin_weight[pin_id];
                pin_grad[pin_id][c] = x_grad * delta_x;
                pin_grad[root_id][c] += root_grad * delta_x;
            }
        }
    }
}

void calc_node_grad_cuda(torch::Tensor node_grad,
                         torch::Tensor pin_id2node_id,
                         torch::Tensor pin_grad,
                         torch::Tensor node2pin_list,
                         torch::Tensor node2pin_list_end,
                         int num_nodes,
                         bool deterministic) {
    if (deterministic) {
        auto stream = at::cuda::getCurrentCUDAStream();
        const int threads = 128;
        const int blocks = (num_nodes * 2 + threads - 1) / threads;
        calc_node_grad_deterministic_cuda_kernel<<<blocks, threads, 0, stream>>>(
            node_grad.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            pin_grad.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
            node2pin_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
            node2pin_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
            num_nodes);
    } else {
        const auto pin_id2node_id_view = pin_id2node_id.unsqueeze(1).expand({-1, 2});
        node_grad.scatter_add_(0, pin_id2node_id_view, pin_grad);
    }
}

std::vector<torch::Tensor> wa_wirelength_timing_weight_cuda(torch::Tensor node_pos,
                                                            torch::Tensor timing_pin_weight,
                                                            torch::Tensor pin_id2node_id,
                                                            torch::Tensor pin_rel_cpos,
                                                            torch::Tensor node2pin_list,
                                                            torch::Tensor node2pin_list_end,
                                                            torch::Tensor hyperedge_list,
                                                            torch::Tensor hyperedge_list_end,
                                                            torch::Tensor net_mask,
                                                            torch::Tensor net_weight,
                                                            torch::Tensor hpwl_scale,
                                                            float gamma,
                                                            bool deterministic) {
    hipSetDevice(node_pos.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const auto num_nodes = node_pos.size(0);
    const auto num_pins = pin_id2node_id.size(0);
    const auto num_nets = hyperedge_list_end.size(0);
    const auto num_channels = 2;  // x, y

    auto pin_pos = pin_rel_cpos.clone();  // pin
    auto partial_wa_wl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto partial_hpwl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto pin_grad = torch::zeros({num_pins, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));

    const int threads = 128;
    const int blocks = (num_pins * 2 + threads - 1) / threads;

    node_pos_to_pin_pos_cuda_kernel<<<blocks, threads, 0, stream>>>(
        node_pos.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        pin_id2node_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        pin_pos.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        num_pins);

    const int threads2 = 128;
    const int blocks2 = (num_nets * 2 + threads2 - 1) / threads2;

    float inv_gamma = 1 / gamma;
    wa_wirelength_pin_root_timing_kernel<<<blocks2, threads2, 0, stream>>>(
        pin_pos.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        timing_pin_weight.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
        hyperedge_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        hyperedge_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        net_mask.packed_accessor32<bool, 1, torch::RestrictPtrTraits>(),
        net_weight.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
        hpwl_scale.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
        partial_wa_wl.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        partial_hpwl.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        pin_grad.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        num_nets,
        inv_gamma);

    auto node_grad = torch::zeros({num_nodes, num_channels}, torch::dtype(pin_grad.dtype()).device(pin_grad.device()));
    calc_node_grad_cuda(
        node_grad, pin_id2node_id, pin_grad, node2pin_list, node2pin_list_end, num_nodes, deterministic);

    return {partial_wa_wl, node_grad, partial_hpwl};
}